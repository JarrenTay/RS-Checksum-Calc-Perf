#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <map>
#include <cstdint>
#include <bitset>
#include <filesystem>
#include <chrono>
#include <iomanip>
#include <sstream>
#include "RSChecksumCalculator.h"

using namespace std;
using namespace chrono;

const long long PID = 1321080;
const int DATA_ORDER_G = 1;
const int DATA_ORDER_A = 4;
const int DATA_ORDER_E = 7;
const int DATA_ORDER_M = 10;
const string CSV_HEADER = "Player frame,Enemy Frame,Player TID/SID,Enemy TID/SID,Species,Held Item,Moves,Pokeball,Egg,Enemy Mon";
const string MATCH_FILE = "gpuMatches.csv";
const string ACE_FILE = "gpuAces.csv";
const int DATA_SIZE = 18;
const int POKEBALL_COUNT = 12;
const int OUTPUT_SIZE = 10;
//entry count 12144 data count 218592 output count 121440
//enemyListSize = 506

struct ChecksumMatchResults {
    bool match;
    bool ace;
    long long keyXorData0;
    long long keyXorData3;
    long long keyXorData4;
    long long keyXorData10;
};

/* ******************************************************
 * Purpose: Calculates checksum based on data array and
 *   player and enemy key
 * ******************************************************
 * Parameters:
 *   data: Array of enemy mon data
 *   playerKey: Player key
 *   enemyKey: Enemy key
 * ******************************************************
*/
__global__
void calculateMatchCuda(long long entries, long long *dataTotal, long long *outputTotal) {
    for (long long entry = 0; entry < entries; entry++) {
        long long originalChecksum = ((dataTotal[(entry * DATA_SIZE) + 0] % 65536) + (dataTotal[(entry * DATA_SIZE) + 0] / 65536) + (dataTotal[(entry * DATA_SIZE) + 1] % 65536) + (dataTotal[(entry * DATA_SIZE) + 1] / 65536) + (dataTotal[(entry * DATA_SIZE) + 2] % 65536)
            + (dataTotal[(entry * DATA_SIZE) + 2] / 65536) + (dataTotal[(entry * DATA_SIZE) + 3] % 65536) + (dataTotal[(entry * DATA_SIZE) + 3] / 65536) + (dataTotal[(entry * DATA_SIZE) + 4] % 65536) + (dataTotal[(entry * DATA_SIZE) + 4] / 65536)
            + (dataTotal[(entry * DATA_SIZE) + 5] % 65536) + (dataTotal[(entry * DATA_SIZE) + 5] / 65536) + (dataTotal[(entry * DATA_SIZE) + 6] % 65536) + (dataTotal[(entry * DATA_SIZE) + 6] / 65536) + (dataTotal[(entry * DATA_SIZE) + 7] % 65536)
            + (dataTotal[(entry * DATA_SIZE) + 7] / 65536) + (dataTotal[(entry * DATA_SIZE) + 8] % 65536) + (dataTotal[(entry * DATA_SIZE) + 8] / 65536) + (dataTotal[(entry * DATA_SIZE) + 9] % 65536) + (dataTotal[(entry * DATA_SIZE) + 9] / 65536)
            + (dataTotal[(entry * DATA_SIZE) + 10] % 65536) + (dataTotal[(entry * DATA_SIZE) + 10] / 65536) + (dataTotal[(entry * DATA_SIZE) + 11] % 65536) + (dataTotal[(entry * DATA_SIZE) + 11] / 65536)) % 65536;

        long long keysXored = dataTotal[(entry * DATA_SIZE) + 12] ^ dataTotal[(entry * DATA_SIZE) + 13];

        long long newChecksum = (((keysXored ^ dataTotal[(entry * DATA_SIZE) + 0]) % 65536) + ((keysXored ^ dataTotal[(entry * DATA_SIZE) + 0]) / 65536) + ((keysXored ^ dataTotal[(entry * DATA_SIZE) + 1]) % 65536) + ((keysXored ^ dataTotal[(entry * DATA_SIZE) + 1]) / 65536) + ((keysXored ^ dataTotal[(entry * DATA_SIZE) + 2]) % 65536)
            + ((keysXored ^ dataTotal[(entry * DATA_SIZE) + 2]) / 65536) + ((keysXored ^ dataTotal[(entry * DATA_SIZE) + 3]) % 65536) + ((keysXored ^ dataTotal[(entry * DATA_SIZE) + 3]) / 65536) + ((keysXored ^ dataTotal[(entry * DATA_SIZE) + 4]) % 65536) + ((keysXored ^ dataTotal[(entry * DATA_SIZE) + 4]) / 65536)
            + ((keysXored ^ dataTotal[(entry * DATA_SIZE) + 5]) % 65536) + ((keysXored ^ dataTotal[(entry * DATA_SIZE) + 5]) / 65536) + ((keysXored ^ dataTotal[(entry * DATA_SIZE) + 6]) % 65536) + ((keysXored ^ dataTotal[(entry * DATA_SIZE) + 6]) / 65536) + ((keysXored ^ dataTotal[(entry * DATA_SIZE) + 7]) % 65536)
            + ((keysXored ^ dataTotal[(entry * DATA_SIZE) + 7]) / 65536) + ((keysXored ^ dataTotal[(entry * DATA_SIZE) + 8]) % 65536) + ((keysXored ^ dataTotal[(entry * DATA_SIZE) + 8]) / 65536) + ((keysXored ^ dataTotal[(entry * DATA_SIZE) + 9]) % 65536) + ((keysXored ^ dataTotal[(entry * DATA_SIZE) + 9]) / 65536)
            + ((keysXored ^ dataTotal[(entry * DATA_SIZE) + 10]) % 65536) + ((keysXored ^ dataTotal[(entry * DATA_SIZE) + 10]) / 65536) + ((keysXored ^ dataTotal[(entry * DATA_SIZE) + 11]) % 65536) + ((keysXored ^ dataTotal[(entry * DATA_SIZE) + 11]) / 65536)) % 65536;

        if (originalChecksum == newChecksum) {
            bool ace = (((keysXored ^ dataTotal[(entry * DATA_SIZE) + 0]) % 65536) == 39710);
            outputTotal[(entry * OUTPUT_SIZE) + 0] = 1;                                               // long long match;
            outputTotal[(entry * OUTPUT_SIZE) + 1] = ace;                                             // long long ace;
            outputTotal[(entry * OUTPUT_SIZE) + 2] = keysXored ^ dataTotal[(entry * DATA_SIZE) + 0];  // long long keyXorData0;
            outputTotal[(entry * OUTPUT_SIZE) + 3] = keysXored ^ dataTotal[(entry * DATA_SIZE) + 3];  // long long keyXorData3;
            outputTotal[(entry * OUTPUT_SIZE) + 4] = keysXored ^ dataTotal[(entry * DATA_SIZE) + 4];  // long long keyXorData4;
            outputTotal[(entry * OUTPUT_SIZE) + 5] = keysXored ^ dataTotal[(entry * DATA_SIZE) + 10]; // long long keyXorData10;
            outputTotal[(entry * OUTPUT_SIZE) + 6] = dataTotal[(entry * DATA_SIZE) + 14];             // long long tid;
            outputTotal[(entry * OUTPUT_SIZE) + 7] = dataTotal[(entry * DATA_SIZE) + 15];             // long long frame;
            outputTotal[(entry * OUTPUT_SIZE) + 8] = dataTotal[(entry * DATA_SIZE) + 16];             // long long pokeballIndex;
            outputTotal[(entry * OUTPUT_SIZE) + 9] = dataTotal[(entry * DATA_SIZE) + 17];             // long long enemyListIndex;

        } else {
            outputTotal[(entry * OUTPUT_SIZE) + 0] = 0;  // long long match;
            outputTotal[(entry * OUTPUT_SIZE) + 1] = 0;  // long long ace;
            outputTotal[(entry * OUTPUT_SIZE) + 2] = 2;  // long long keyXorData0;
            outputTotal[(entry * OUTPUT_SIZE) + 3] = 3;  // long long keyXorData3;
            outputTotal[(entry * OUTPUT_SIZE) + 4] = 4;  // long long keyXorData4;
            outputTotal[(entry * OUTPUT_SIZE) + 5] = 5;  // long long keyXorData10;
            outputTotal[(entry * OUTPUT_SIZE) + 6] = 6;  // long long tid;
            outputTotal[(entry * OUTPUT_SIZE) + 7] = 7;  // long long frame;
            outputTotal[(entry * OUTPUT_SIZE) + 8] = 8;  // long long pokeballIndex;
            outputTotal[(entry * OUTPUT_SIZE) + 9] = 9;  // long long enemyListIndex;
        }
    }
}

int main(int argc, char* argv[]) {
    steady_clock::time_point start = steady_clock::now();
    
    // Argument Parsing
    vector<int> arguments = parseArguments(argc, argv);
    handleArguments(arguments);

    // Parse Data Files
    vector<string> enemyList = {};
    map<string, vector<long long>> enemyDict = dataFileToMap("enemyDataList.csv", enemyList);
    vector<vector<int>> otidVector = otidFileToVector("OTIDs.csv");
    string dataOrder[24] = {
        "GAEM", "GAME", "GEAM", "GEMA", "GMAE", "GMEA",
        "AGEM", "AGME", "AEGM", "AEMG", "AMGE", "AMEG",
        "EGAM", "EGMA", "EAGM", "EAMG", "EMGA", "EMAG",
        "MGAE", "MGEA", "MAGE", "MAEG", "MEGA", "MEAG"
    };

    // Calculate Checksums
    cout << "Executing with TIDs " << arguments[0] << " to " << arguments[1] << " (inclusive) and the first " << arguments[2] << " frames." << endl;
    cout << "USING CUDA" << endl;
    calculateChecksumMatches(arguments[0], arguments[1], arguments[2], dataOrder, enemyList, enemyDict, otidVector);

    // Check Time Elapsed
    steady_clock::time_point end = steady_clock::now();
    cout << "Time elapsed: " << (duration_cast<microseconds> (end - start).count()) / 1000000 << " seconds" << std::endl;
    return 0;
}

/* ******************************************************
 * Purpose: Parses passed arguments and assigns defaults
 * ******************************************************
 * Parameters:
 *   argc: Number of arguments
 *   argv: Char* array of arguments
 * ******************************************************
*/
vector<int> parseArguments(int argc, char* argv[]) {
    vector<int> arguments = vector<int>();
    if (argc >= 2) {
        string arg = argv[1];
        int startingTid = stoi(arg);
        arguments.push_back(startingTid);
    } else {
        arguments.push_back(3575);
    }
    if (argc >= 3) {
        string arg = argv[2];
        int endingTid = stoi(arg);
        arguments.push_back(endingTid);
    } else {
        arguments.push_back(3575);
    }
    if (argc >= 4) {
        string arg = argv[3];
        int frameCount = stoi(arg);
        arguments.push_back(frameCount);
    } else {
        arguments.push_back(4000);
    }
    return arguments;
}

/* ******************************************************
 * Purpose: Checks for tid and frame argument validity
 * ******************************************************
 * Parameters:
 *   args: Vector [TID start, TID end, Frame amount]
 * ******************************************************
*/
void handleArguments(vector<int> &args) {
    if (args[0] < 0) {
        cout << "TID lower bound exceeded, set to 0." << endl;
        args[0] = 0;
    }
    if (args[0] > 100000) {
        cout << "TID upper bound exceeded, set to 100000." << endl;
        args[0] = 100000;
    }
    if (args[1] < args[0]) {
        cout << "TID range error, upper bound set to lower bound." << endl;
        args[1] = args[0];
    }
    if (args[1] > 100000) {
        cout << "TID upper bound exceeded, set to 100000." << endl;
        args[1] = 100000;
    }
    if (args[2] < 1) {
        cout << "Frame lower bound exceeded, set to 1." << endl;
        args[2] = 1;
    }
    if (args[2] > 100000) {
        cout << "Frame upper bound exceeded, set to 100000." << endl;
        args[2] = 100000;
    }
}

/* ******************************************************
 * Purpose: Parses enemyDataList.csv
 * ******************************************************
 * Parameters:
 *   fileName: Name of file containing enemy data
 *   enemyList: Outputs vector of enemy mons
 * ******************************************************
*/
map<string, vector<long long>> dataFileToMap(string fileName, vector<string> &enemyList) {
    string enemyDataRawLine = "";
    ifstream enemyDataFile(fileName);
    map<string, vector<long long>> enemyDict = map<string, vector<long long>>();
    const string DELIMITER = ",";

    while (getline(enemyDataFile, enemyDataRawLine)) {
        string enemyMon = enemyDataRawLine.substr(0, enemyDataRawLine.find(DELIMITER));
        string enemyData = enemyDataRawLine.substr(enemyDataRawLine.find(DELIMITER) + 1);
        vector<long long> enemyDataVector = vector<long long>();
        
        long long pieceOne = hexStringToIntLittleEndian(enemyData.substr(0, 8));
        enemyDataVector.push_back(pieceOne);
        for (int i = 0; i < 12; i++) {
            int pieceTwoStart = (i + 8) * 8;
            long long pieceTwo = hexStringToIntLittleEndian(enemyData.substr(pieceTwoStart, 8));
            long long pieceThree = hexStringToIntLittleEndian(enemyData.substr(8 , 8));
            enemyDataVector.push_back(pieceTwo ^ pieceOne ^ pieceThree);
        }
        enemyDict.insert(pair<string, vector<long long>>(enemyMon, enemyDataVector));
        enemyList.push_back(enemyMon);
    }
    return enemyDict;
}

/* ******************************************************
 * Purpose: Converts a hex string to little endian
 * ******************************************************
 * Parameters:
 *   hexString: Hex string to convert
 * ******************************************************
*/
long long hexStringToIntLittleEndian(string hexString) {
    string reversedHexString = "";
    for (int i = 0; i * 2 < hexString.length(); i++) {
        reversedHexString = hexString.substr(i * 2, 2) + reversedHexString;
    }
    return stoll(reversedHexString, 0, 16);
}

/* ******************************************************
 * Purpose: Parses Otid file into a 2d array
 * ******************************************************
 * Parameters:
 *   fileName: Hex string to convert
 * ******************************************************
*/
vector<vector<int>> otidFileToVector(string fileName) {
    string otidDataRawLine = "";
    ifstream otidDataFile(fileName);
    vector<vector<int>> otidVector = vector<vector<int>>();
    const string DELIMITER = ",";
    bool headerRow = true;

    while (getline(otidDataFile, otidDataRawLine)) {
        if (headerRow) {
            headerRow = false;
            continue;
        }

        int firstCommaIndex = otidDataRawLine.find(DELIMITER);
        int secondCommaIndex = otidDataRawLine.find(DELIMITER, firstCommaIndex + 1);
        int thirdCommaIndex = otidDataRawLine.find(DELIMITER, secondCommaIndex + 1);
        string advancesString = otidDataRawLine.substr(0, firstCommaIndex);
        string tidString = otidDataRawLine.substr(firstCommaIndex + 1, secondCommaIndex);
        string sidString = otidDataRawLine.substr(secondCommaIndex + 1, thirdCommaIndex);
        vector<int> otidRow = vector<int>();
        
        otidRow.push_back(stoi(advancesString));
        otidRow.push_back(stoi(tidString));
        otidRow.push_back(stoi(sidString));

        otidVector.push_back(otidRow);
    }
    return otidVector;
}

/* ******************************************************
 * Purpose: Loops through TIDs and frames and calcs
 *   checksums for each combination.
 * ******************************************************
 * Parameters:
 *   trainerIdStart: first TID to calc
 *   trainerIdEnd: last TID to calc
 *   frames: num frames to calc
 *   dataOrder: idk what this is
 *   enemyList: vector of enemy mons
 *   enemyDict: map of enemy mon to enemy data
 *   otidVector: vector of otid data
 * ******************************************************
*/
void calculateChecksumMatches(int trainerIdStart, int trainerIdEnd, int frames, string dataOrder[], vector<string> &enemyList, map<string, vector<long long>> enemyDict, vector<vector<int>> otidVector) {

    // Delete output files if they exist and create a new one.
    try {
        filesystem::remove(MATCH_FILE);
        filesystem::remove(ACE_FILE);   
    } catch (int errorCode) { }
    ofstream matchFile(MATCH_FILE);
    ofstream aceFile(ACE_FILE);
    matchFile << CSV_HEADER << endl;
    aceFile << CSV_HEADER << endl;

    // Trainer ID is inclusive. We don't do subtraction in TID like in python bc we don't need to account for header row.
    for (int tid = trainerIdStart; tid <= trainerIdEnd; tid++) {
        cout << "Checking tid " << tid << endl;

        string playerHex = intToHex(otidVector[tid][2], 4) + intToHex(otidVector[tid][1], 4).substr(2);
        long long playerLongLong = stoll(playerHex, 0, 16);
        long long playerKey = PID ^ playerLongLong;
        long long enemyListSize = enemyList.size();
        long long entryCount = frames * enemyListSize * POKEBALL_COUNT;
        long long dataCount = entryCount * DATA_SIZE;
        long long outputCount = entryCount * OUTPUT_SIZE;
        //long long *dataTotal = new long long[dataCount];
        //long long *outputTotal = new long long[outputCount];
        long long *dataTotal, *outputTotal;
        hipMallocManaged(&dataTotal, dataCount * sizeof(long long));
        hipMallocManaged(&outputTotal, outputCount * sizeof(long long));
		cout << "entry count " << entryCount << " data count " << dataCount << " output count " << outputCount << endl;
        
        long long *data = new long long[DATA_SIZE];

        // Start at frame 0. Python version starts at 1 bc of header column
        for (int frame = 0; frame < frames; frame++) {
            if (frame % 1 == 0) {
                cout << "Checking frame " << frame << endl;
            }

            string enemyHex = intToHex(otidVector[frame][1], 4) + intToHex(otidVector[frame][2], 4).substr(2);
            long long enemyLongLong = stoll(enemyHex, 0, 16);
            long long enemyKey = PID ^ enemyLongLong;

            data[12] = playerKey;
            data[13] = enemyKey;
            data[14] = tid;
            data[15] = frame;

            // Loop through all mons
            for (int enemyListIndex = 0; enemyListIndex < enemyListSize; enemyListIndex++) {
                string enemyMon = enemyList[enemyListIndex];
                vector<long long> enemyMonData = enemyDict[enemyMon];
                string dataOrderString = dataOrder[enemyMonData[0] % 24];

                for (int dataOrderCharIndex = 0; dataOrderCharIndex < dataOrderString.length(); dataOrderCharIndex++) {
                    int enemyMonIndex = 0;
                    switch (dataOrderString[dataOrderCharIndex]) {
                        case 'G':
                            enemyMonIndex = DATA_ORDER_G;
                            break;
                        case 'A':
                            enemyMonIndex = DATA_ORDER_A;
                            break;
                        case 'E':
                            enemyMonIndex = DATA_ORDER_E;
                            break;
                        case 'M':
                            enemyMonIndex = DATA_ORDER_M;
                            break;
                    }
                    data[dataOrderCharIndex * 3] = enemyMonData[enemyMonIndex];
                    data[(dataOrderCharIndex * 3) + 1] = enemyMonData[enemyMonIndex + 1];
                    data[(dataOrderCharIndex * 3) + 2] = enemyMonData[enemyMonIndex + 2];
                }
                
                data[17] = enemyListIndex;

                // Loop through pokeballs. We quit as soon as we find a match, even though there are likely more of the same pokeball.
                for (int pokeballIndex = 1; pokeballIndex < 13; pokeballIndex++) {
                    data[9] = stoll(llToBin(data[9], 32).substr(2, 1) + llToBin(pokeballIndex, 4).substr(2) + llToBin(data[9], 32).substr(7), 0, 2);
                    data[16] = pokeballIndex;

                    for (int dataIndex = 0; dataIndex < DATA_SIZE; dataIndex++) {
                        //cout << "Adding to " << (frame * enemyListSize * POKEBALL_COUNT * DATA_SIZE) + (enemyListIndex * POKEBALL_COUNT * DATA_SIZE) + ((pokeballIndex - 1) * DATA_SIZE) + dataIndex << endl;
                        dataTotal[(frame * enemyListSize * POKEBALL_COUNT * DATA_SIZE) + (enemyListIndex * POKEBALL_COUNT * DATA_SIZE) + ((pokeballIndex - 1) * DATA_SIZE) + dataIndex] = data[dataIndex];
                    }

                    /*
                    // [data[0], data[1], data[2], data[3], data[4], data[5], data[6], data[7], data[8], data[9], data[10], data[11], playerKey, enemyKey, tid, frame, pokeballIndex, enemyListIndex]
                    for (int i = 0; i < DATA_SIZE; i++) {
                        cout << data[i] << " ";
                    }
                    cout << endl;*/
                }
            }
        }
        /*
        for (long long entry = 0; entry < entryCount; entry++) {
            cout << "data entry: " << entry << " ";
            //const string CSV_HEADER = "Player frame,Enemy Frame,Player TID/SID,Enemy TID/SID,Species,Held Item,Moves,Pokeball,Egg,Enemy Mon";
            cout << dataTotal[(entry * DATA_SIZE) + 0] << " " << dataTotal[(entry * DATA_SIZE) + 1] << " "
                << dataTotal[(entry * DATA_SIZE) + 2] << " " << dataTotal[(entry * DATA_SIZE) + 3] << " "
                << dataTotal[(entry * DATA_SIZE) + 4] << " " << dataTotal[(entry * DATA_SIZE) + 5] << " "
                << dataTotal[(entry * DATA_SIZE) + 6] << " " << dataTotal[(entry * DATA_SIZE) + 7] << " "
                << dataTotal[(entry * DATA_SIZE) + 8] << " " << dataTotal[(entry * DATA_SIZE) + 9] << " "
                << dataTotal[(entry * DATA_SIZE) + 10] << " " << dataTotal[(entry * DATA_SIZE) + 11] << " "
                << dataTotal[(entry * DATA_SIZE) + 12] << " " << dataTotal[(entry * DATA_SIZE) + 13] << " "
                << dataTotal[(entry * DATA_SIZE) + 14] << " " << dataTotal[(entry * DATA_SIZE) + 15] << " "
                << dataTotal[(entry * DATA_SIZE) + 16] << " " << dataTotal[(entry * DATA_SIZE) + 17] << " " << endl;
        }*/

        calculateMatchCuda<<<1, 1>>>(entryCount, dataTotal, outputTotal);
        //calculateMatchCuda(entryCount, dataTotal, outputTotal);

        hipDeviceSynchronize();
        cout << "finished calcing checksums" << endl;
		cout << "entryCount: " << entryCount << endl;
        for (long long entry = 0; entry < entryCount; entry++) {
            /*cout << "entry: " << entry;
            //const string CSV_HEADER = "Player frame,Enemy Frame,Player TID/SID,Enemy TID/SID,Species,Held Item,Moves,Pokeball,Egg,Enemy Mon";
			cout << outputTotal[(entry * OUTPUT_SIZE) + 0] << " " << outputTotal[(entry * OUTPUT_SIZE) + 1] << " "
				<< outputTotal[(entry * OUTPUT_SIZE) + 2] << " " << outputTotal[(entry * OUTPUT_SIZE) + 3] << " "
				<< outputTotal[(entry * OUTPUT_SIZE) + 4] << " " << outputTotal[(entry * OUTPUT_SIZE) + 5] << " "
				<< outputTotal[(entry * OUTPUT_SIZE) + 6] << " " << outputTotal[(entry * OUTPUT_SIZE) + 7] << " "
				<< outputTotal[(entry * OUTPUT_SIZE) + 8] << " " << outputTotal[(entry * OUTPUT_SIZE) + 9] << endl;
            */
            
            if (outputTotal[(entry * OUTPUT_SIZE) + 0]) {
                string matchOut =
                    to_string(tid) + "," +
                    to_string(outputTotal[(entry * OUTPUT_SIZE) + 7]) + "," +
                    to_string(otidVector[tid][1]) + " " +
                    to_string(otidVector[tid][2]) + "," +
                    to_string(otidVector[outputTotal[(entry * OUTPUT_SIZE) + 7]][2]) + " " +
                    to_string(otidVector[outputTotal[(entry * OUTPUT_SIZE) + 7]][1]) + "," +
                    "0x" + intToHex(outputTotal[(entry * OUTPUT_SIZE) + 2], 8).substr(6) + "," +
                    intToHex(outputTotal[(entry * OUTPUT_SIZE) + 2], 8).substr(0, 6) + "," +
                    "0x" + intToHex(outputTotal[(entry * OUTPUT_SIZE) + 3], 8).substr(6) + " " +
                    intToHex(outputTotal[(entry * OUTPUT_SIZE) + 3], 8).substr(0, 6) + " " +
                    "0x" + intToHex(outputTotal[(entry * OUTPUT_SIZE) + 4], 8).substr(6) + " " +
                    intToHex(outputTotal[(entry * OUTPUT_SIZE) + 4], 8).substr(0, 6) + "," +
                    to_string(outputTotal[(entry * OUTPUT_SIZE) + 8]) + "," +
                    llToBin(outputTotal[(entry * OUTPUT_SIZE) + 5], 32).substr(3, 1) + "," +
                    enemyList[outputTotal[(entry * OUTPUT_SIZE) + 9]];
                matchFile << matchOut << endl;

                if (outputTotal[(entry * OUTPUT_SIZE) + 1]) {
                    aceFile << matchOut << endl;
                }
            }
                
        }

        hipFree(dataTotal);
        hipFree(outputTotal);
        delete [] data;
    }

    matchFile.close();
    aceFile.close();
}

/* ******************************************************
 * Purpose: Calculates checksum based on data array and
 *   player and enemy key
 * ******************************************************
 * Parameters:
 *   data: Array of enemy mon data
 *   playerKey: Player key
 *   enemyKey: Enemy key
 * ******************************************************
*/
ChecksumMatchResults calculateMatch(long long data[], long long playerKey, long long enemyKey) {

    long long originalChecksum = ((data[0] % 65536) + (data[0] / 65536) + (data[1] % 65536) + (data[1] / 65536) + (data[2] % 65536)
        + (data[2] / 65536) + (data[3] % 65536) + (data[3] / 65536) + (data[4] % 65536) + (data[4] / 65536)
        + (data[5] % 65536) + (data[5] / 65536) + (data[6] % 65536) + (data[6] / 65536) + (data[7] % 65536)
        + (data[7] / 65536) + (data[8] % 65536) + (data[8] / 65536) + (data[9] % 65536) + (data[9] / 65536)
        + (data[10] % 65536) + (data[10] / 65536) + (data[11] % 65536) + (data[11] / 65536)) % 65536;

    long long keysXored = playerKey ^ enemyKey;

    long long newChecksum = (((keysXored ^ data[0]) % 65536) + ((keysXored ^ data[0]) / 65536) + ((keysXored ^ data[1]) % 65536) + ((keysXored ^ data[1]) / 65536) + ((keysXored ^ data[2]) % 65536)
        + ((keysXored ^ data[2]) / 65536) + ((keysXored ^ data[3]) % 65536) + ((keysXored ^ data[3]) / 65536) + ((keysXored ^ data[4]) % 65536) + ((keysXored ^ data[4]) / 65536)
        + ((keysXored ^ data[5]) % 65536) + ((keysXored ^ data[5]) / 65536) + ((keysXored ^ data[6]) % 65536) + ((keysXored ^ data[6]) / 65536) + ((keysXored ^ data[7]) % 65536)
        + ((keysXored ^ data[7]) / 65536) + ((keysXored ^ data[8]) % 65536) + ((keysXored ^ data[8]) / 65536) + ((keysXored ^ data[9]) % 65536) + ((keysXored ^ data[9]) / 65536)
        + ((keysXored ^ data[10]) % 65536) + ((keysXored ^ data[10]) / 65536) + ((keysXored ^ data[11]) % 65536) + ((keysXored ^ data[11]) / 65536)) % 65536;

    if (originalChecksum == newChecksum) {
        bool ace = (((keysXored ^ data[0]) % 65536) == 39710);
        ChecksumMatchResults matchResults = {
            true,                   // bool match;
            ace,                    // bool ace;
            keysXored ^ data[0],    // long long keyXorData0;
            keysXored ^ data[3],    // long long keyXorData3;
            keysXored ^ data[4],    // long long keyXorData4;
            keysXored ^ data[10],   // long long keyXorData10;
        };

        return matchResults;
    } else {
        ChecksumMatchResults matchResults = {
            false,  // bool match;
            false,  // bool ace;
            0,      // long long keyXorData0;
            0,      // long long keyXorData3;
            0,      // long long keyXorData4;
            0,      // long long keyXorData10;
        };

        return matchResults;
    }
}

/* ******************************************************
 * Purpose: Replaces spaces in a number string with 0
 * ******************************************************
 * Parameters:
 *   number: string to pad
 * ******************************************************
*/
string padStringNumber(string number) {
    string outNumber = number;
    for (int charIndex = 0; charIndex < number.length(); charIndex++) {
        if (number[charIndex] == ' ') {
            outNumber[charIndex] = '0';
        }
    }
    return outNumber;
}

template< typename T >
string intToHex(T i, int len)
{
    stringstream stream;
    stream << std::setfill('0') << std::setw(sizeof(T) * 2)
        << std::hex << i;

    string hex = stream.str();
    return "0x" + hex.substr(hex.length() - len);
}

string llToBin(long long longlong, int len)
{
    return "0b" + bitset<32>(longlong).to_string().substr(32 - len);
}